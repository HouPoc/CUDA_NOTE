#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define LEN 20000



int main(){
    int inputSize = sizeof(float) * LEN;
    int numBlocks = ceil((float)LEN/(float)COMMON_WIDTH);
    dim3 grid = (numBlocks, 1, 1);
    dim3 block = (COMMON_WIDTH, 1, 1);

    float *h_input = (float*)malloc(inputSize);
    float *h_output = (float *)malloc(sizeof(float) * numBlocks);
    float *d_input;
    float *d_output;
    
    initial2DMatrix<float>(h_input, 1, LEN, 1);
    

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_input, inputSize);
    hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);

    hipMalloc(&d_output, sizeof(float)* numBlocks);

    hipEventRecord(start);
    sumReduction<<<grid, block>>>(float* input, float *output, int len, int stride);
    hipEventRecord(stop);

    hipMemcpy(h_input, d_input, inputSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_output, d_output, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    
    hipFree(d_input);
    free(h_input);

}