#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define LEN 400000

void sumReduction(){
    
}

int main(){
    enum RD {   
                reductionWithBranch = 1,
                reductionWithNoBranch = 2,
                reductionWithBranchNoIdleThread = 3,
                reductionWithNoBranchNoIdleThread = 4
            };
    RD reductionStrategy = reductionWithNoBranchNoIdleThread; 

   
    int numBlocks;
    int inputSize = sizeof(float) * LEN;
    if (reductionStrategy == reductionWithBranch 
     || reductionStrategy == reductionWithNoBranch)
    {
        numBlocks = ceil((float)LEN/(float)COMMON_WIDTH);
          
    }
    if (reductionStrategy == reductionWithBranchNoIdleThread 
     || reductionStrategy == reductionWithNoBranchNoIdleThread)
    {
        numBlocks = ceil((float)LEN/(float)(2 * COMMON_WIDTH));   
    }
    dim3 block(COMMON_WIDTH, 1, 1);
    dim3 grid(numBlocks, 1, 1); 
    
    float ms;

    float *h_input = (float*)malloc(inputSize);
    float *h_output = (float *)malloc(sizeof(float) * numBlocks);

    float *d_input;
    float *d_output;
    
    initial2DMatrix<float>(h_input, 1, LEN, 1);
    

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_input, inputSize);
    hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);

    hipMalloc(&d_output, sizeof(float)* numBlocks);


    hipEventRecord(start);
    switch (reductionStrategy){
        case reductionWithBranch: 
            printf("reductionWithBranch \n");
            sumReduction<<<grid, block>>>(d_input, d_output, LEN); break;
        case reductionWithNoBranch:
            printf("reductionWithNoBranch \n");
            sumReductionNoBranch<<<grid, block>>>(d_input, d_output, LEN); break;
        case reductionWithBranchNoIdleThread:
            printf("reductionWithBranchNoIdleThread \n");
            sumReductionModify<<<grid, block>>>(d_input, d_output, LEN); break;
        case reductionWithNoBranchNoIdleThread: 
            printf("reductionWithNoBranchNoIdleThread \n");
            sumReductionNoBranchModify<<<grid, block>>>(d_input, d_output, LEN); break;
    }
    hipEventRecord(stop);

    hipMemcpy(h_input, d_input, inputSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_output, d_output, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    checkSumReduction<float>(h_input, h_output, LEN, numBlocks);
    hipFree(d_input);
    free(h_input);

}