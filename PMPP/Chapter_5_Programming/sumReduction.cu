#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define LEN 200000



int main(){
    int inputSize = sizeof(float) * LEN;
    int numBlocks = ceil((float)LEN/(float)COMMON_WIDTH);
    float ms;
    dim3 grid(numBlocks, 1, 1);
    dim3 gridModifyOne(ceil(numBlocks/2), 1, 1);
    dim3 block(COMMON_WIDTH, 1, 1);

    float *h_input = (float*)malloc(inputSize);
    float *h_output = (float *)malloc(sizeof(float) * numBlocks);
    float *d_input;
    float *d_output;
    
    initial2DMatrix<float>(h_input, 1, LEN, 1);
    

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_input, inputSize);
    hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);

    hipMalloc(&d_output, sizeof(float)* numBlocks);

    hipEventRecord(start);
    //sumReduction<<<grid, block>>>(d_input, d_output, LEN);
    //sumReductionNoBranch<<<grid, block>>>(d_input, d_output, LEN);
    sumReductionModifyOne<<<gridModifyOne, block>>>(d_input, d_output, LEN);
    hipEventRecord(stop);

    hipMemcpy(h_input, d_input, inputSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_output, d_output, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    checkSumReduction<float>(h_input, h_output, LEN, numBlocks);
    hipFree(d_input);
    free(h_input);

}