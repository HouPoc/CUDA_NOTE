#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"


int main(){
    int sizeMatrixLeft = sizeof(float) * ROW_LEFT * K;
    int sizeMatrixRight = sizeof(float) * K * COL_RIGHT;
    int sizeMatrixOutput = sizeof(float) * ROW_LEFT * COL_RIGHT; 
    float *h_matrixLeft     =   (float *) malloc(sizeMatrixLeft);
    float *h_matrixRight    =   (float *) malloc(sizeMatrixRight);
    float *h_matrixOutput         =   (float *) malloc(sizeMatrixOutput);
    initial2DMatrix<float>(h_matrixLeft, ROW_LEFT, K, 0);
    initial2DMatrix<float>(h_matrixRight, K, COL_RIGHT, 0);
    float *d_matrixLeft, *d_matrixRight, *d_matrixOutput;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    err = hipMalloc(&d_matrixLeft, sizeMatrixLeft);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_matrixLeft, h_matrixLeft, sizeMatrixLeft, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_matrixRight, sizeMatrixRight);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_matrixRight, h_matrixRight, sizeMatrixRight, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_matrixOutput, sizeMatrixOutput);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    /* 
        Invoke Simple MatrixMult Kernel
    */
    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(ceil((float)COL_RIGHT/(float)(2 *TILE_WIDTH)), ceil((float)ROW_LEFT/(float)TILE_WIDTH));
    hipEventRecord(start);
    matrixMultTiledIncreasedGranularity<<<grid, block>>>(d_matrixLeft, d_matrixRight, d_matrixOutput);
    //MatrixMatrixMult<<<grid, block>>>(d_matrixLeft, d_matrixRight, d_matrixOutput);
    hipEventRecord(stop);
    err = hipMemcpy(h_matrixOutput, d_matrixOutput, sizeMatrixOutput, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    hipFree(d_matrixLeft);
    hipFree(d_matrixRight);
    hipFree(d_matrixOutput);
    checkMatrixMatrixMult<float>(h_matrixLeft, h_matrixRight, h_matrixOutput, ROW_LEFT, K, K, COL_RIGHT);
    free(h_matrixLeft);
    free(h_matrixRight);
    free(h_matrixOutput);
}