#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define ROW_LEFT 512 
#define COL_RIGHT 256
#define K 1000

int main(){
    int sizeMatrixLeft = sizeof(float) * ROW_LEFT * K;
    int sizeMatrixRight = sizeof(float) * K * COL_RIGHT;
    int sizeMatrixOutput = sizeof(float) * ROW_LEFT * COL_RIGHT; 
    float *h_matrixLeft     =   (float *) malloc(sizeMatrixLeft);
    float *h_matrixRight    =   (float *) malloc(sizeMatrixRight);
    float *h_matrixOutput         =   (float *) malloc(sizeMatrixOutput);
    initial2DMatrix<float>(h_matrixLeft, ROW_LEFT, COL_RIGHT, 1);
    initial2DMatrix<float>(h_matrixRight, ROW_LEFT, COL_RIGHT, 1);
    float *d_matrixLeft, *d_matrixRight, *d_matrixOutput;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    err = hipMalloc(&d_matrixLeft, sizeMatrixLeft);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_matrixLeft, h_matrixLeft, sizeMatrixLeft, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_matrixRight, sizeMatrixRight);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_matrixRight, h_matrixRight, sizeMatrixRight, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_matrixOutput, sizeMatrixOutput);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    /* 
        Invoke Simple MatrixMult Kernel
    */
    dim3 grid(ceil(ROW_LEFT/32), ceil(COL_RIGHT/32), 1);
    dim3 block(32, 32 ,1);
    hipEventRecord(start);
    MatrixMatrixMultTiled<<<grid, block>>>(d_matrixLeft, d_matrixRight, d_matrixOutput);
    //MatrixMatrixMult<<<grid, block>>>(d_matrixLeft, d_matrixRight, d_matrixOutput);
    hipEventRecord(stop);
    err = hipMemcpy(h_matrixOutput, d_matrixOutput, sizeMatrixOutput, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    hipFree(d_matrixLeft);
    hipFree(d_matrixRight);
    hipFree(d_matrixOutput);
    checkMatrixMatrixMult<float>(h_matrixLeft, h_matrixRight, h_matrixOutput, ROW_LEFT, K, K, COL_RIGHT);
    free(h_matrixLeft);
    free(h_matrixRight);
    free(h_matrixOutput);
}