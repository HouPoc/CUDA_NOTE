#include "hip/hip_runtime.h"
#include "../cuda_debug.h"
#include "../cuda_note.h"

#define DATA_DIM 1000

int main() {
    int h_row = DATA_DIM;
    int h_col = DATA_DIM;
    size_t maskSize = sizeof(float) * MASK_WIDTH * MASK_WIDTH;
    size_t dataSize = sizeof(float) * DATA_DIM * DATA_DIM;
    float *h_mask2D = (float*)malloc(maskSize);
    float *h_input = (float*) malloc(dataSize);
    float *h_output = (float*) malloc(dataSize);
    initial2DMatrix<float>(h_input, DATA_DIM, DATA_DIM, 1);
    initial2DMatrix<float>(h_mask2D, MASK_WIDTH, MASK_WIDTH, 1);
    float *d_input, *d_output;

    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    hipMemcpy(d_input, h_input, dataSize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask2D), h_mask2D, maskSize);
 
    dim3 block(I_TILE_SIZE, I_TILE_SIZE, 1);

    dim3 grid(ceil(DATA_DIM/(float)O_TILE_SIZE), ceil(DATA_DIM/(float)O_TILE_SIZE), 1);
    
    convolution2D<<<grid, block>>>(d_input, d_output, h_row, h_col, (h_col + MASK_WIDTH - 1), MASK_WIDTH);
    
    std :: cout << hipGetErrorString(hipGetLastError()) << std :: endl;
    hipMemcpy(h_output, d_output, dataSize, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    check2Dconvolution<float>(h_input, h_mask2D, h_output, h_row, h_col, MASK_WIDTH, MASK_WIDTH);
    free(h_input);
    free(h_output);
    free(h_mask2D);

    return 0;
}