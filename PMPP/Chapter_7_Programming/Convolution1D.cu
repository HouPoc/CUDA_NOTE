#include "hip/hip_runtime.h"
#include <iostream>
#include "../cuda_debug.h"
#include "../cuda_note.h"


#define LEN 2000    // define the size of input/output data
#define BLOCK_DIM 256


int main(){
    size_t maskSize = sizeof(float) * MASK_WIDTH;
    size_t dataSize = sizeof(float) * LEN;
    float *mask =   (float *) malloc(maskSize);
    float *h_input =  (float *) malloc(dataSize);
    float *h_output = (float *) malloc(dataSize);
    initial2DMatrix<float>(h_input,1, LEN, 0);
    initial2DMatrix<float>(mask, 1, MASK_WIDTH, 0);
    float *d_input, *d_output;
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);
    hipMemcpy(d_input, h_input, dataSize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask, maskSize);

    dim3 block(BLOCK_DIM);
    dim3 grid(ceil(LEN/(float)BLOCK_DIM));
    hipEventRecord(start);
    convolution1D<<<grid, block>>>(d_input, d_output, LEN, MASK_WIDTH);
    hipEventSynchronize(stop);
    hipMemcpy(h_output, d_output, dataSize, hipMemcpyDeviceToHost);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time is %f ms \n",  ms);
    hipFree(d_input);
    hipFree(d_output);
    check1Dconvolution<float>(h_input, mask, h_output, LEN, MASK_WIDTH);
    free(h_input);
    free(h_output);
    free(mask);

    return 0;
}