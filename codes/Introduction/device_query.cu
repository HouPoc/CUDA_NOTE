#include<iostream>
#include"../cuda_note.h"

int main(){
  int dev_count = 0;
  hipError_t err = hipGetDeviceCount(&dev_count);
  if (err == hipSuccess)
  {
    platformInfo(&dev_count);
  }
  return 0;
}
