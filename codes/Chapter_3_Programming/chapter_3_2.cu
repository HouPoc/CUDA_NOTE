#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define ROW 1024
#define COL 1024

int main(){
    int sizeMatrix = sizeof(float) * ROW * COL;
    int sizeVector = sizeof(float) * ROW;
    float *h_matrixInput    =   (float *) malloc(sizeMatrix);
    float *h_vectorInput    =   (float *) malloc(sizeVector);
    float *h_output         =   (float *) malloc(sizeVector);
    float *d_matrixInput, *d_vectorInput, *d_output;
    initial2DMatrix<float>(h_matrixInput, ROW, COL);
    initial2DMatrix<float>(h_vectorInput, 1, COL);
    hipMalloc(&d_matrixInput, sizeMatrix);
    hipMalloc(&d_vectorInput, sizeVector);
    hipMalloc(&d_output, sizeVector);
    hipMemcpy(d_matrixInput, h_matrixInput, sizeMatrix, hipMemcpyHostToDevice);
    hipMemcpy(d_vectorInput, h_vectorInput, sizeVector, hipMemcpyHostToDevice);
    
    matrixVectorMult<<<ceil(ROW/256), 256>>>(d_matrixInput, d_vectorInput, d_output, COL, ROW);
    hipMemcpy(h_output, d_output, sizeVector, hipMemcpyDeviceToHost);
    hipFree(d_matrixInput);
    hipFree(d_vectorInput);
    hipFree(d_output);
    
    checkMatrixVectorMult<float>(h_matrixInput, h_vectorInput, h_output, ROW, COL);

}