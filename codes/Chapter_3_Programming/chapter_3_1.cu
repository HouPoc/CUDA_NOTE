#include "hip/hip_runtime.h"
#include<iostream>
#include"../cuda_note.h"
#include"../cuda_debug.h"
#define ROW 1024
#define COL 1024

/*Chapter 3 Problem 1*/
int main(int argc, char* argv[]){
    int size = sizeof(float) * ROW * COL;
    float *h_firstInput     =   (float *) malloc(size);
    float *h_secondInput    =   (float *) malloc(size);
    float *h_output         =   (float *) malloc(size);
    initial2DMatrix<float>(h_firstInput, ROW, COL, 1);
    initial2DMatrix<float>(h_secondInput, ROW, COL, 1);
    float *d_firstInput, *d_secondInput, *d_output;
    hipError_t err;
    int debug = 0;
    if (debug){ 
      peakMatrix<float> (h_firstInput, ROW, COL);
      peakMatrix<float> (h_secondInput, ROW, COL);
    } 
    
    
    err = hipMalloc(&d_firstInput, size);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_firstInput, h_firstInput, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_secondInput, size);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMemcpy(d_secondInput, h_secondInput, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc(&d_output, size);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    /*Chapter 3 problem 1.a --> matrixAdd*/
    //matrixAdd <<<ceil(ROW * COL/512), 512>>> (d_firstInput, d_secondInput, d_output, ROW * COL);
    /*Chapter 3 problem 1.b --> matrixAddRow*/
    //matrixAddRow <<<ceil(ROW/ 512), 512>>> (d_firstInput, d_secondInput, d_output, ROW * COL, COL);
    /*Chapter 3 problem 1.c --> matrixAddCol*/
    matrixAddRow <<<ceil(COL/ 512), 512>>> (d_firstInput, d_secondInput, d_output, ROW * COL, ROW);

    err = hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    hipFree(d_firstInput);
    hipFree(d_secondInput);
    hipFree(d_output);
    checkAdditionResult<float>(h_firstInput, h_secondInput, h_output, ROW, COL);
    free(h_firstInput);
    free(h_secondInput);
    free(h_output);
}